#include "hip/hip_runtime.h"
/*
 * Bins fast calculation for later convolution.
 * Takes a gpuArray input and returns a gpuArray output
 * e.g.       bin_num_q           = BinNumCalZ(q);
 * or        [bin_num_q,bin_num_x]= BinNumCalZ(q,x);
 * [bin_num_q,bin_num_x,bin_num_y]= BinNumCalZ(q,x,y);
 * Author: Tianlong He. 
 * Time  : 20200519
 */
 
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "hip/hip_runtime.h"
/*
 * Device code
 */
void __global__ BinNumCal(float const * const Z,
                         float * const binZ,
                         int const N,
						 int const rowsZ,
						 int const binnum)
{
    /* Calculate the global linear index, assuming a 1-d grid. */
	int i;
	int index,indexz;
	int const iL = threadIdx.x * N;
	int const iR = iL + N;
	extern __shared__ float binZshare[];
	
	for (i=0;i<binnum;i++) {
		binZshare[threadIdx.x * binnum + i] = 0;
	}
	/*__syncthreads();*/
	for (i=iL;i<iR;i+=5) {
		
		if (i < rowsZ) {
			indexz = blockIdx.x * rowsZ + i;
			index = (int)(Z[indexz]);
			binZshare[threadIdx.x*binnum+index] += 1;
			index = (int)(Z[indexz + 1]);
			binZshare[threadIdx.x*binnum+index] += 1;
			index = (int)(Z[indexz + 2]);
			binZshare[threadIdx.x*binnum+index] += 1;
			index = (int)(Z[indexz + 3]);
			binZshare[threadIdx.x*binnum+index] += 1;
			index = (int)(Z[indexz + 4]);
			binZshare[threadIdx.x*binnum+index] += 1;
		}
	}
	/*__syncthreads();*/
	for (i=0;i<binnum;i++) {
		index= (i + blockIdx.x * binnum)*blockDim.x + threadIdx.x;
		binZ[index] = binZshare[threadIdx.x * binnum + i];
	}
	
}

/*
 * Host code
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    /* Declare all variables.*/
    mxGPUArray const *Z;
    mxGPUArray *binZ;
	float const *d_Z;
    float *d_binZ;
	size_t binnum;
    int N;
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file.";

    /* Choose a reasonably sized number of threads for the block. */
    int const threadsPerBlock = 5;
    int blocksPerGrid;

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();

    /* Throw an error if the input is not a GPU array. */
    if ((nrhs!=2) || !(mxIsGPUArray(prhs[1]))) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }
	
	binnum = (int)(mxGetScalar(prhs[0]));
    Z = mxGPUCreateFromMxArray(prhs[1]);

    /*
     * Verify that Z really is a double array before extracting the pointer.
     */
    if (mxGPUGetClassID(Z) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    /* 
     * Now that we have verified the data type, extract a pointer to the input
     * data on the device.
     */
    d_Z = (float const *)(mxGPUGetDataReadOnly(Z));
	
	const mwSize *dimsZ = mxGPUGetDimensions(Z);
	size_t nrowsZ = dimsZ[0];
	size_t ncolsZ = dimsZ[1];	
			
	mwSize dims[2] = {threadsPerBlock, ncolsZ * binnum}; // note here	
	
    /* Create a GPUArray to hold the result and get its underlying pointer. */
    binZ = mxGPUCreateGPUArray(2,
                            dims,
                            mxGPUGetClassID(Z),
                            mxGPUGetComplexity(Z),
                            MX_GPU_INITIALIZE_VALUES);
							
    d_binZ = (float *)(mxGPUGetData(binZ));

    /*
     * Call the kernel using the CUDA runtime API. We are using a 1-d grid here,
     * and it would be possible for the number of elements to be too large for
     * the grid. For this example we are not guarding against this possibility.
     */
    N = ((int)(nrowsZ)+threadsPerBlock-1)/threadsPerBlock;
	
    blocksPerGrid = (int)(ncolsZ);
    BinNumCal<<<blocksPerGrid, threadsPerBlock, threadsPerBlock*binnum*sizeof(float)>>>(d_Z, d_binZ, N, nrowsZ, binnum);

    /* Wrap the result up as a MATLAB gpuArray for return. */
    plhs[0] = mxGPUCreateMxArrayOnGPU(binZ);

    /*
     * The mxGPUArray pointers are host-side structures that refer to device
     * data. These must be destroyed before leaving the MEX function.
     */
    mxGPUDestroyGPUArray(Z);
    mxGPUDestroyGPUArray(binZ);
}
 